#include "hip/hip_runtime.h"
/*
 * render.cu
 *
 *  Created on: Mar 6, 2013
 *      Author: norton
 */

/* local includes */
#include <render.hpp>
#include <Debug.hpp>

/* boost includes */
#include <boost/variant.hpp>

/* std includes */
#include <float.h>
#include <vector>

/* include the vector code */
#include <Vector.cui>
//#include <thrust/functional.h>

#define MAXIMUM_ITERATIONS   512
#define MINIMUM_CONTRIBUTION 0.0039

namespace ray {

  namespace render {

    /* ********************************************************************** */
    /* *** Forward Function declarations ************************************ */
    /* ********************************************************************** */

    template<typename T>
    struct d_Stack {
        __device__ d_Stack() :
            stack(), top(0) { }

        __device__ inline void push(T elem) { stack[top++] = elem;   }
        __device__ inline T&   peek()       { return stack[top - 1]; }
        __device__ inline T     pop()       { return stack[--top];   }
        __device__ inline uint size() const { return top;            }

        T    stack[128];
        uint top;
    };

    struct selem {
        enum state { left = 0, right = 1, done = 2};

        __device__ selem() :
          surf(-1), lr(done) { }
        __device__ selem(int32_t surf) :
          surf(surf), lr(left) { }

        int32_t surf;
        state lr;
    };

    struct {
        uint x;
    } threadIdx;

    __host__ std::ostream& operator<<(std::ostream& ostr, d_Intersection& inter) {
      return (ostr << "INTER[ l:" << inter.location
                        << ", n:" << inter.normal
                        << ", v:" << inter.viewing << "]");
    }

    __host__ std::ostream& operator<<(std::ostream& ostr, d_Ray& ray) {
      return (ostr << "RAY[ L:" << ray.L << ", U:" << ray.U << "]");
    }

    /**
     * Structure used to simplify passing of arguments for the device functions.
     * All of the pointers to relevant arrays are found here instead of passing
     * each individually.
     */
    struct d_Model {
        uint32_t root;
        uint32_t n_lights;

        d_Surface*  surfaces;
        d_Material* materials;
        d_Light*    lights;
    };

    __device__ d_Intersection best_of(
        const d_Intersection& a,
        const d_Intersection& b);

    __device__ bool intersect(
        Vector m,
        Vector l,
        const d_Ray& ray);

    __device__ bool intersect(
        d_Model* surfs,
        uint32_t curr,
        const d_Ray& ray,
        d_Intersection& inter);

    __device__ bool intersect(
        d_Surface& curr,
        const d_Ray& ray,
        d_Intersection& inter);

    __device__ Vector normalAt(
        d_Surface& surf, const
        Vector& inter);

    __device__ Vector diffuse(
        const Vector* lhs,
        const Vector& rhs);

    __device__ bool shadowed(
        d_Model* model,
        const d_Ray& ray,
        const d_Light& light);

    __device__ Vector reflectance(
        d_Model* model,
        const d_Intersection& inter);

    __device__ Vector getColor(
        d_Model* model,
        const d_Ray& ray);

    /* ********************************************************************** */
    /* *** Intersection Code ************************************************ */
    /* ********************************************************************** */

    /**
     * Find the best of two different intersections. This compares the distance
     * of the intersections and picks the closer one.
     *
     * @param a  the first intersection
     * @param b  the second intersection
     * @return   the best of the two intersections
     */
    __device__ d_Intersection best_of(
        const d_Intersection& a,
        const d_Intersection& b)
    {
      return a.distance < b.distance ?
          a.distance > 0 ? a : b :
          b.distance > 0 ? b : a;
    }

    /**
     * Determines if a ray and a bounding box intersect. This is used by every
     * surface before the actual intersection is run. This allows us to short
     * circuit and ignore large sections of the model on every test.
     *
     * @param m    the minimum coordinates of the bounding box.
     * @param l    the length of the sides of a bounding box.
     * @param ray  the ray that we are performing the intersection for
     * @return     if the ray passes through the region represented by the box
     */
    __device__ bool intersect(
        Vector m,
        Vector l,
        const d_Ray& ray)
    {
      double tmin, tmax;
      double dmin, dmax;

      if(ray.zero(0)) {
        if(ray.posi(0)) {
          dmin = (m.x() - ray.L.x()) * ray.iU.x();
          dmax = dmin + (l.x() * ray.iU.x());
          if(dmax < EPSILON)
            return false;
        } else {
          dmax = (m.x() - ray.L.x()) * ray.iU.x();
          if(dmax < EPSILON)
            return false;
          dmin = dmax + (l.x() * ray.iU.x());
        }

        if(dmin > dmax)
          return false;
      } else {
        if((ray.L.x() < m.x()) || (ray.L.x() > l.x() + m.x())) {
          return false;
        }

        dmin = FLT_MIN;
        dmax = FLT_MAX;
      }

      if(ray.zero(1)) {
        if(ray.posi(1)) {
          tmin = (m.y() - ray.L.y()) * ray.iU.y();
          tmax = tmin + (l.y() * ray.iU.y());
        } else {
          tmax = (m.y() - ray.L.y()) * ray.iU.y();
          tmin = tmax + (l.y() * ray.iU.y());
        }

        if(tmax < dmax) {
          if(tmax < EPSILON)
            return false;
          if(tmin > dmin) {
            if(tmin > tmax)
              return false;
            dmin = tmin;
          } else if(dmin > tmax) {
            return false;
          }
          dmax = tmax;
        } else {
          if(tmin > dmin) {
            if(tmin > dmax)
              return false;
            dmin = tmin;
          }
        }
      } else {
        if((ray.L.y() < m.y()) || (ray.L.y() > l.y() + m.y())) {
          return false;
        }
      }

      if(ray.zero(2)) {
        if(ray.posi(2)) {
          tmin = (m.z() - ray.L.z()) * ray.iU.z();
          tmax = tmin + (l.z() * ray.iU.z());
        } else {
          tmax = (m.z() - ray.L.z()) * ray.iU.z();
          tmin = tmax + (l.z() * ray.iU.z());
        }

        if(tmax < dmax) {
          if(tmax < EPSILON)
            return false;
          if(tmin > dmin) {
            if(tmin > tmax)
              return false;
            dmin = tmin;
          } else if(dmin > tmax) {
            return false;
          }
          dmax = tmax;
        } else {
          if(tmin > dmin) {
            if(tmin > dmax)
              return false;
            dmin = tmin;
          }
        }
      } else {
        if((ray.L.z() < m.z()) || (ray.L.z() > l.z() + m.z())) {
          return false;
        }
      }

      return true;
    }

    /**
     * Intersects a ray with a generic surface. This will check if the ray
     * passes through the region that contains the surface and then calls the
     * appropriate function to determine if it intersects the actual surface.
     *
     * @param surfs  The array of all surfaces in the model
     * @param curr   The index of the current surface.
     * @param ray    The ray to find the intersection for.
     * @param inter  Return for the location of intersection
     * @return       If the ray intersected the surface
     */
    __device__ bool intersect(
        d_Model* model,
        uint32_t root,
        const d_Ray& ray,
        d_Intersection& inter)
    {
      d_Intersection best;
      d_Intersection curr;

      d_Stack<selem> stack;

      bool found = false;

      stack.push(selem(root));

      while(stack.size() != 0) {
        if(stack.peek().surf >= 0) {
          d_Surface& surf = model->surfaces[stack.peek().surf];

          if(!intersect(surf.min, surf.len, ray) ||
              ray.src == surf.id) {
            stack.pop();
          } else if(surf.which == d_Surface::triangle) {
            if(intersect(surf, ray, curr)) {
              best = best_of(best, curr);
              found = true;
            }
            stack.pop();

          } else if(surf.which == d_Surface::tree) {
            switch(stack.peek().lr) {
              case selem::left:
                stack.peek().lr = selem::right;
                stack.push(selem(surf.d_axis));
                break;
              case selem::right:
                stack.peek().lr = selem::done;
                stack.push(selem(surf.v_axis));
                break;
              case selem::done:
                stack.pop();
                break;
            }
          }
        } else {
          stack.pop();
        }
      }

      if(found) {
        inter = best;
        return true;
      }
      return false;
    }

    /**
     * Intersections a ray with a Triangle. This is the base case for the
     * recursive intersection check.
     *
     * @param curr   The triangle that will be checked for intersection
     * @param ray    The ray that will be intersected
     * @param inter  Return for the location of intersection
     * @return       If the ray intersected the surface
     */
    __device__ bool intersect(
        d_Surface& curr,
        const d_Ray& ray,
        d_Intersection& inter)
    {
      double  a, b, uu, uv, vv, wu, wv, D, gamma, beta, r;
      Vector w, I;
      Vector u, v, n;

      u = curr.vb - curr.va;
      v = curr.vc - curr.va;
      n = cross(u, v);

      a = -dot(n, ray.L - curr.va);
      b =  dot(n, ray.U);
      if(fabs(b) < EPSILON) {
        return false;
      }

      if((r = a / b) < 0.0) {
        return false;
      }

      I  = ray.L + (ray.U * r);
      uu = dot(u, u);
      uv = dot(u, v);
      vv = dot(v, v);
      w  = I - curr.va;
      wu = dot(w, u);
      wv = dot(w, v);
      D  = uv * uv - uu * vv;

      gamma = (uv * wv - vv * wu) / D;
      if(gamma < 0.0 || gamma > 1.0) {
        return false;
      }

      beta = (uv * wu - uu * wv) / D;
      if(beta < 0.0 || (gamma + beta) > 1.0) {
        return false;
      }

      inter = d_Intersection(curr.id, I, normalAt(curr, I), ray.U.normalize(), r);
      return true;
    }

    /**
     * Finds the normal for the location intersection of a ray and a Triangle.
     *
     * @param surf   The triangle that the ray intersected
     * @param inter  The location of the intersection
     * @return       The normal for the location of intersection
     */
    __device__ Vector normalAt(
        d_Surface& surf,
        const Vector& inter)
    {
      double  u, v;
      Vector diff, nt1, nt2;

      u    = dot((inter - surf.va), surf._perp) / dot(surf.vc - surf.va, surf._perp);
      diff = surf.va + ((inter - surf.va) / u);
      v    = ((diff[surf.v_axis] - surf.vb[surf.v_axis]) /
              (surf.vc[surf.v_axis] - surf.vb[surf.v_axis]));

      nt1 = surf.na + ((surf.nb - surf.na) * u);
      nt2 = surf.na + ((surf.nc - surf.na) * u);

      return (nt1 + ((nt2 - nt1) * v)).normalize();
    }

    /* ********************************************************************** */
    /* *** Reflectance Code ************************************************* */
    /* ********************************************************************** */

    /**
     * Operator used for the calculation of the diffuse reflectance off of a
     * material.
     *
     * @param lhs  the matrix that contains the diffuse reflectance
     * @param rhs  the color of the light that is being reflected
     * @return     the color that reflects off the surface
     */
    __device__ Vector diffuse(
        const Vector* lhs,
        const Vector& rhs)
    {
      return Vector(
          lhs[0][0] * rhs[0] + lhs[0][1] * rhs[0] + lhs[0][2] * rhs[0],
          lhs[1][0] * rhs[1] + lhs[1][1] * rhs[1] + lhs[1][2] * rhs[1],
          lhs[2][0] * rhs[2] + lhs[2][1] * rhs[2] + lhs[2][2] * rhs[2]);
    }

    /**
     * Test if a particular location is shadowed.
     *
     * @param model  the Model that needs to be checks for shadows
     * @param ray    the ray from the location to the light
     * @param light  the light source that is being checked
     * @return       true if the location is shadowed.
     */
    __device__ bool shadowed(
        d_Model* model,
        const d_Ray& ray,
        const d_Light& light)
    {
      d_Intersection inter;

      double maxDistance = light.local.distance(ray.L);
      return(
          intersect(model, model->root, ray, inter) &&
          inter.distance < maxDistance);
    }

    /**
     * Get color of the light at an intersection.
     *
     * @param model  the model to get the color for
     * @param inter  the location to get the color for
     * @return       the color at that location
     */
    __device__ Vector reflectance(
        d_Model* model,
        const d_Intersection& inter)
    {
      d_Material& m = model->materials[model->surfaces[inter.src].mat];

      Vector p = inter.location;
      Vector v = inter.viewing.negate();
      Vector n = inter.normal;

      Vector Lp;
      Vector Rl;

      Vector ret;

      if(dot(v, n) < 0)
        n = n.negate();

      for(int i = 0; i < model->n_lights; i++) {
        const d_Light& light = model->lights[i];

        Lp = (light.local - p).normalize();

        if(dot(Lp, n) < 0 && shadowed(model, d_Ray(Lp, p, inter.src), light))
          continue;

        Rl = (n * (dot(Lp, n) * 2) - Lp).normalize();

        ret = ret +
            (diffuse(m.diffuse, light.illum) * dot(Lp, n)) +
            (light.illum * m.ks * pow(max(0.0, dot(v, Rl)), m.alpha));
      }

      return ret;
    }

    /**
     * Given a source Ray and a model, this gets the color that we should use
     * when filling in the pixel.
     *
     * @param model  the model to render
     * @param ray    the source ray from the camera
     * @return       the color that the pixel should be filled with
     */
    __device__ Vector getColor(
        d_Model* model,
        const d_Ray& ray)
    {
      d_Intersection inter;
      Vector         color, newdir;
      Vector         n, v;
      d_Ray          curr_ray = ray;
      double          cont = 1.0;

      for(int i = 0; i < MAXIMUM_ITERATIONS && cont > MINIMUM_CONTRIBUTION; i++) {
        if(!intersect(model, model->root, curr_ray, inter))
          break;

        v = inter.viewing.negate();
        n = inter.normal;

        color = color + (reflectance(model, inter) * cont);
        cont  = cont * (model->materials[model->surfaces[inter.src].mat].ks);

        newdir = n * (dot(v, n) * 2) - v;
        curr_ray = d_Ray(inter.location, newdir.normalize(), inter.src);
      }

      return ray::max(ray::min(color, 255), 0);
    }

    /* ********************************************************************** */
    /* *** External Interface *********************************************** */
    /* ********************************************************************** */

    __global__ void kernel(
        d_Surface*  surfaces,
        d_Material* materials,
        d_Light*    lights,
        uint32_t    root,
        uint32_t    n_lights,
        d_Ray*      rays,
        Vector*     dest)
    {
      d_Model model;

      model.root      = root;
      model.n_lights  = n_lights;
      model.surfaces  = surfaces;
      model.materials = materials;
      model.lights    = lights;

      uint idx = threadIdx.x;

      dest[idx] = getColor(&model, rays[idx]);
    }

    d_Surface*  surfaces  = NULL;
    d_Material* materials = NULL;
    d_Light*    lights    = NULL;

    uint32_t rootSurface;

    int32_t n_surface;
    int32_t n_material;
    int32_t n_light;

    __host__ void setSurfaces(d_Surface* surs, size_t size, uint32_t root) {
      size_t total = size * sizeof(d_Surface);

      if(surfaces)
        hipFree(surfaces);

      hipMalloc((void**)&surfaces, total);
      hipMemcpy(surfaces, surs, total, hipMemcpyHostToDevice);

      rootSurface = root;
      n_surface = size;
    }

    __host__ void setMaterials(d_Material* mats, size_t size) {
      size_t total = size * sizeof(d_Material);

      if(materials)
        hipFree(materials);

      hipMalloc((void**)&materials, total);
      hipMemcpy(materials, mats, total, hipMemcpyHostToDevice);

      n_material = size;
    }

    __host__ void setLights(d_Light* ligs, size_t size) {
      size_t total = size * sizeof(d_Light);

      if(lights)
        hipFree(lights);

      hipMalloc((void**)&lights, total);
      hipMemcpy(lights, ligs, total, hipMemcpyHostToDevice);

      n_light = size;
    }

    __host__ void Trace(Vector* out, d_Ray* in, size_t size) {
      Vector* device_out;
      d_Ray*  device_in;

      hipMalloc((void**)&device_out, size * sizeof(Vector));
      hipMalloc((void**)&device_in,  size * sizeof(d_Ray));

      hipMemcpy(device_in, in, size * sizeof(d_Ray), hipMemcpyHostToDevice);

#ifdef __HIPCC__
      kernel<<<1, 256>>>(
          surfaces,
          materials,
          lights,
          rootSurface,
          n_light,
          device_in,
          device_out);
#else
      for(int i = 0; i < size; i++) {
        threadIdx.x = i;
        kernel(
            surfaces,
            materials,
            lights,
            rootSurface,
            n_light,
            device_in,
            device_out);
      }
#endif

      hipMemcpy(out, device_out, size * sizeof(Vector), hipMemcpyDeviceToHost);

      hipFree(device_out);
      hipFree(device_in);
    }

    __host__ std::ostream& operator<<(std::ostream& ostr, const d_Surface& surf) {
      ostr << surf.id << " => ";

      switch(surf.which) {
        case d_Surface::triangle:
          ostr << "Triangle";
          break;

        case d_Surface::tree:
          ostr << "Tree: " << surf.v_axis << " :: " << surf.d_axis;
          break;
      }

      return ostr;
    }

  }

}
